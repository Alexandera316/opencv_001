#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>

#include <iostream>

#include <hip/hip_runtime.h>

#include "image_processing.hpp"
#include "opencv2/core/cuda_types.hpp"

#define checkCudaErrors(val) check( (val), #val, __FILE__, __LINE__)

template<typename T>
void check(T err, const char* const func, const char* const file, const int line) {
	if (err != hipSuccess)
	{
		std::stringstream errorStream;
		errorStream << "CUDA error at: " << file << ":" << line << "\n";
		errorStream << hipGetErrorString(err) << " " << func << "\n";
		throw std::runtime_error( errorStream.str() );
	}
}


__global__
void convertImageSimple(
	cv::cuda::PtrStepSz<uchar4> _source,
	cv::cuda::PtrStepSz<uchar4> _target,
	unsigned char _threshold
)
{
	int y = blockDim.y*blockIdx.y + threadIdx.y;
	int x = blockDim.x*blockIdx.x + threadIdx.x;

	if (x >= _source.cols || y >= _target.rows)
		return;

	const uchar4& sourcePixel = _source(y, x);
	uchar4& targetPixel = _target(y, x);

	unsigned char pixelValue =
		( sourcePixel.x * 307 +
		  sourcePixel.y * 604 +
		  sourcePixel.z * 113 ) >> 10;

	targetPixel.w = sourcePixel.w;
	targetPixel.x = targetPixel.y = targetPixel.z =
		pixelValue > _threshold ? 255 : 0;
}

#define WARP_SIZE 32

__global__
void convertImageBlocks(
	cv::cuda::PtrStepSz<uchar4> _source,
	cv::cuda::PtrStepSz<uchar4> _target,
	int _elemsPerThread,
	unsigned char _threshold
)
{
	int y = blockDim.y * blockIdx.y + threadIdx.y;
	int x = blockDim.x * blockIdx.x + threadIdx.x;

	const int loopStart = (x / WARP_SIZE * WARP_SIZE) * (_elemsPerThread - 1) + x;
	for (int i = loopStart, j = 0; j < _elemsPerThread && i < _source.cols; i += WARP_SIZE, ++j)
	{
		const uchar4& sourcePixel = _source(y, i);
		uchar4& targetPixel = _target(y, i);

		unsigned char pixelValue =
			( sourcePixel.x * 307 +
			  sourcePixel.y * 604 +
			  sourcePixel.z * 113 ) >> 10;

		targetPixel.w = sourcePixel.w;
		targetPixel.x = targetPixel.y = targetPixel.z =
			pixelValue > _threshold ? 255 : 0;
	}
}

void ExecuteThresholdSimpleCUDA(
	cv::cuda::GpuMat& _source,
	cv::cuda::GpuMat& _target,
	unsigned char _threshold
)
{
	hipSetDevice(0);

	dim3 blockSize;
	dim3 gridSize;
	int threadNum;
	
	cv::cuda::PtrStepSz<uchar4> source = _source;
	cv::cuda::PtrStepSz<uchar4> target = _target;

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	threadNum = 512;
	blockSize = dim3(threadNum, 1, 1);
	gridSize = dim3(_source.cols / threadNum + 1, _source.rows, 1);

	hipEventRecord(start);

	convertImageSimple<<<gridSize, blockSize>>> (source, target, _threshold);

	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipDeviceSynchronize(); 
	checkCudaErrors(hipGetLastError());
}

void ExecuteThresholdBlocksCUDA(
	cv::cuda::GpuMat& _source,
	cv::cuda::GpuMat& _target,
	unsigned char _threshold
)
{
	hipSetDevice(0);

	dim3 blockSize;
	dim3 gridSize;
	
	cv::cuda::PtrStepSz<uchar4> source = _source;
	cv::cuda::PtrStepSz<uchar4> target = _target;

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	const int elemsPerThread = 16;
	const int threadNum = 256 ;
	blockSize = dim3(threadNum, 1, 1);
	gridSize = dim3(_source.cols / ( elemsPerThread * threadNum ) + 1, _source.rows, 1);

	hipEventRecord(start);

	convertImageBlocks<<<gridSize, blockSize>>> (source, target, elemsPerThread, _threshold);

	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipDeviceSynchronize(); 
	checkCudaErrors(hipGetLastError());
}
