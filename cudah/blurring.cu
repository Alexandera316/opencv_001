#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>
#include <hip/device_functions.h>
#include <hip/hip_runtime_api.h>
#include <iostream>

#include "image_processing.hpp"
#include "opencv2/core/cuda_types.hpp"

#define checkCudaErrors(val) check( (val), #val, __FILE__, __LINE__)

template<typename T>
void check(T err, const char* const func, const char* const file, const int line) {
	if (err != hipSuccess)
	{
		std::stringstream errorStream;
		errorStream << "CUDA error at: " << file << ":" << line << "\n";
		errorStream << hipGetErrorString(err) << " " << func << "\n";
		throw std::runtime_error(errorStream.str());
	}
}


__global__
void gaussian_blur(
	cv::cuda::PtrStepSz<uchar4> _source,
	cv::cuda::PtrStepSz<uchar4> _target,
	const float * const d_filter,
	const int filterWidth
)
{
	const int row = blockIdx.y * blockDim.y + threadIdx.y;
	const int col = blockIdx.x * blockDim.x + threadIdx.x;

	if (col >= _source.cols || row >= _source.rows)
		return;

	const int halfWidth = filterWidth / 2;

	extern __shared__ float shared_filter[];
	if (threadIdx.y < filterWidth && threadIdx.x < filterWidth)
	{
		const int filterOff = threadIdx.y*filterWidth + threadIdx.x;
		shared_filter[filterOff] = d_filter[filterOff];
	}
	__syncthreads();

	float resR = 0.0f, resG = 0.0f, resB = 0.0f;
	for (int filterRow = -halfWidth; filterRow <= halfWidth; ++filterRow)
	{
		for (int filterCol = -halfWidth; filterCol <= halfWidth; ++filterCol)
		{
			const int imageRow = min(max(row + filterRow, 0), _source.rows - 1);
			const int imageCol = min(max(col + filterCol, 0), _source.cols - 1);

			const float filterValue = shared_filter[(filterRow + halfWidth)*filterWidth + filterCol + halfWidth];
			const uchar4& imagePixel = _source( imageRow, imageCol );

			resR += imagePixel.x * filterValue;
			resG += imagePixel.y * filterValue;
			resB += imagePixel.z * filterValue;
		}
	}

	_target( row, col ) = make_uchar4(resR, resG, resB, 255);
}


void ExecuteGaussianBlur(
	cv::cuda::GpuMat& _source,
	cv::cuda::GpuMat& _target,
	float* _filter,
	int _filterSize
)
{
	hipSetDevice(0);
	checkCudaErrors(hipGetLastError());

	float *d_filter;
	const size_t filterSize = sizeof(float) * _filterSize * _filterSize;
	checkCudaErrors(hipMalloc(&d_filter, filterSize));
	checkCudaErrors(hipMemcpy(d_filter, _filter, filterSize, hipMemcpyHostToDevice));

	cv::cuda::PtrStepSz<uchar4> source = _source;
	cv::cuda::PtrStepSz<uchar4> target = _target;

	dim3 blockSize;
	dim3 gridSize;
	int threadNum;

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	threadNum = 32;
	blockSize = dim3(threadNum, threadNum, 1);
	gridSize = dim3(source.cols / threadNum + 1, source.rows / threadNum + 1, 1);

	hipEventRecord(start);

	gaussian_blur <<<gridSize, blockSize, filterSize>>>(
			_source,
			_target,
			d_filter,
			_filterSize
		);

	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

	checkCudaErrors(hipFree(d_filter));
}
