#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>

#include <iostream>

#include <hip/hip_runtime.h>

#include "image_processing.hpp"
#include "opencv2/core/cuda_types.hpp"

#define checkCudaErrors(val) check( (val), #val, __FILE__, __LINE__)

template<typename T>
void check(T err, const char* const func, const char* const file, const int line) {
	if (err != hipSuccess)
	{
		std::stringstream errorStream;
		errorStream << "CUDA error at: " << file << ":" << line << "\n";
		errorStream << hipGetErrorString(err) << " " << func << "\n";
		throw std::runtime_error( errorStream.str() );
	}
}



__global__
void grayscale_simple(
	cv::cuda::PtrStepSz<uchar4> _source,
	cv::cuda::PtrStepSz<uchar4> _target
)
{
	int y = blockDim.y*blockIdx.y + threadIdx.y;
	int x = blockDim.x*blockIdx.x + threadIdx.x;

	if (x >= _source.cols || y >= _target.rows)
		return;

	const uchar4& sourcePixel = _source(y, x);
	uchar4& targetPixel = _target(y, x);

	targetPixel.w = sourcePixel.w;
	targetPixel.x = targetPixel.y = targetPixel.z =
		( sourcePixel.x * 307 +
		  sourcePixel.y * 604 +
		  sourcePixel.z * 113 ) >> 10;
}

void ExecuteGrayscaleCUDASimple(
	cv::cuda::GpuMat& _source,
	cv::cuda::GpuMat& _target
)
{
	hipSetDevice(0);

	dim3 blockSize;
	dim3 gridSize;
	int threadNum;
	
	cv::cuda::PtrStepSz<uchar4> source = _source;
	cv::cuda::PtrStepSz<uchar4> target = _target;

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	threadNum = 512;
	blockSize = dim3(threadNum, 1, 1);
	gridSize = dim3(_source.cols / threadNum + 1, _source.rows, 1);

	hipEventRecord(start);

	grayscale_simple <<<gridSize, blockSize>>> (source, target);

	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipDeviceSynchronize(); 
	checkCudaErrors(hipGetLastError());
}

void ExecuteGrayscaleCUDAOptimized(
	cv::cuda::GpuMat& _source,
	cv::cuda::GpuMat& _target
)
{
	hipSetDevice(0);

	dim3 blockSize;
	dim3 gridSize;
	
	cv::cuda::PtrStepSz<uchar4> source = _source;
	cv::cuda::PtrStepSz<uchar4> target = _target;

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	const int elemsPerThread = 16;
	const int threadNum = 256 ;
	blockSize = dim3(threadNum, 1, 1);
	gridSize = dim3(_source.cols / ( elemsPerThread * threadNum ) + 1, _source.rows, 1);

	hipEventRecord(start);

	grayscale_optimized <<<gridSize, blockSize>>> (source, target, elemsPerThread);

	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipDeviceSynchronize(); 
	checkCudaErrors(hipGetLastError());

}