#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>

#include <iostream>

#include <hip/hip_runtime.h>

#include "image_processing.hpp"
#include "opencv2/core/cuda_types.hpp"

#define checkCudaErrors(val) check( (val), #val, __FILE__, __LINE__)

template<typename T>
void check(T err, const char* const func, const char* const file, const int line) {
	if (err != hipSuccess)
	{
		std::stringstream errorStream;
		errorStream << "CUDA error at: " << file << ":" << line << "\n";
		errorStream << hipGetErrorString(err) << " " << func << "\n";
		throw std::runtime_error( errorStream.str() );
	}
}

#define WARP_SIZE 32

__global__
void grayscale_optimized(
	cv::cuda::PtrStepSz<uchar4> _source,
	cv::cuda::PtrStepSz<uchar4> _target,
	int _elemsPerThread
)
{
	int y = blockDim.y * blockIdx.y + threadIdx.y;
	int x = blockDim.x * blockIdx.x + threadIdx.x;

	const int loopStart = (x / WARP_SIZE * WARP_SIZE) * (_elemsPerThread - 1) + x;
	for (int i = loopStart, j = 0; j < _elemsPerThread && i < _source.cols; i += WARP_SIZE, ++j)
	{
		const uchar4& sourcePixel = _source(y, i);
		uchar4& targetPixel = _target(y, i);

		targetPixel.w = sourcePixel.w;
		targetPixel.x = targetPixel.y = targetPixel.z =
			( sourcePixel.x * 307 +
			  sourcePixel.y * 604 +
			  sourcePixel.z * 113 ) >> 10;
	}
}

__global__
void grayscale_simple(
	cv::cuda::PtrStepSz<uchar4> _source,
	cv::cuda::PtrStepSz<uchar4> _target
)
{
	int y = blockDim.y*blockIdx.y + threadIdx.y;
	int x = blockDim.x*blockIdx.x + threadIdx.x;

	if (x >= _source.cols || y >= _target.rows)
		return;

	const uchar4& sourcePixel = _source(y, x);
	uchar4& targetPixel = _target(y, x);

	targetPixel.w = sourcePixel.w;
	targetPixel.x = targetPixel.y = targetPixel.z =
		( sourcePixel.x * 307 +
		  sourcePixel.y * 604 +
		  sourcePixel.z * 113 ) >> 10;
}

void ExecuteGrayscaleCUDASimple(
	cv::cuda::GpuMat& _source,
	cv::cuda::GpuMat& _target
)
{
	hipSetDevice(0);

	dim3 blockSize;
	dim3 gridSize;
	int threadNum;
	
	cv::cuda::PtrStepSz<uchar4> source = _source;
	cv::cuda::PtrStepSz<uchar4> target = _target;

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	threadNum = 512;
	blockSize = dim3(threadNum, 1, 1);
	gridSize = dim3(_source.cols / threadNum + 1, _source.rows, 1);

	hipEventRecord(start);

	grayscale_simple <<<gridSize, blockSize>>> (source, target);

	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipDeviceSynchronize(); 
	checkCudaErrors(hipGetLastError());
}

void ExecuteGrayscaleCUDAOptimized(
	cv::cuda::GpuMat& _source,
	cv::cuda::GpuMat& _target
)
{
	hipSetDevice(0);

	dim3 blockSize;
	dim3 gridSize;
	
	cv::cuda::PtrStepSz<uchar4> source = _source;
	cv::cuda::PtrStepSz<uchar4> target = _target;

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	const int elemsPerThread = 16;
	const int threadNum = 256 ;
	blockSize = dim3(threadNum, 1, 1);
	gridSize = dim3(_source.cols / ( elemsPerThread * threadNum ) + 1, _source.rows, 1);

	hipEventRecord(start);

	grayscale_optimized <<<gridSize, blockSize>>> (source, target, elemsPerThread);

	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipDeviceSynchronize(); 
	checkCudaErrors(hipGetLastError());

}